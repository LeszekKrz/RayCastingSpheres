#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <glad/glad.h>
#include <GLFW/glfw3.h>

#include "shader.hpp"
#include "functions.hpp"
#include "cudaFunctions.cuh"

#include <stdio.h>
#include <iostream>


// settings
const unsigned int SCR_WIDTH = 800;
const unsigned int SCR_HEIGHT = 600;

void framebuffer_size_callback(GLFWwindow* window, int width, int height);
void processInput(GLFWwindow* window);

hipError_t fillWithCuda(unsigned char* table, int SCR_WIDTH, int SCR_HEIGHT);

__global__ void fillKernel(char* table, int width)
{
    char* m_table = table + blockIdx.x * width * 3 + blockIdx.y * 3;
    *m_table = 0;
    *(m_table + 1) = 255;
    *(m_table + 2) = 0;
}
__global__ void fillKernel2(char* table, int width)
{
    char* m_table = table + blockIdx.x * width * 3 + blockIdx.y * 3;
    *m_table = 0;
    *(m_table + 1) = 0;
    *(m_table + 2) = 255;
}

int main()
{
    // glfw: initialize and configure
    // ------------------------------
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    int size = SCR_WIDTH * SCR_HEIGHT;

    unsigned char* h_texture;
    unsigned char* d_texture;
    h_texture = (unsigned char*)malloc(size * 3);
    PrepareTexture(&d_texture, size * 3);

    fillWithCuda(h_texture, SCR_WIDTH, SCR_HEIGHT);

    circles h_circles, d_circles;
    h_circles.n = 1000;
    CreateCircles(&h_circles);
    PrepareCircles(h_circles, &d_circles);
    DisplayCircles(h_circles);

    lights h_lights, d_lights;
    h_lights.n = 3;
    CreateLights(&h_lights);
    PrepareLights(h_lights, &d_lights);
    DisplayLights(h_lights);

    scene d_scene{ d_circles, d_lights };
    


    // glfw window creation
    // --------------------
    GLFWwindow* window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "LearnOpenGL", NULL, NULL);
    if (window == NULL)
    {
        std::cout << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

    // glad: load all OpenGL function pointers
    // ---------------------------------------
    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return -1;
    }


    Shader ourShader("VertexShader.txt", "FragmentShader.txt");

    // set up vertex data (and buffer(s)) and configure vertex attributes
    // ------------------------------------------------------------------
    //float vertices[] = {
    //    // positions         // colors
    //     1.0f, 1.0f, 0.0f,  1.0f, 0.0f, 0.0f, 1.0f, 1.0f,  
    //     1.0f, -1.0f, 0.0f,  0.0f, 1.0f, 0.0f, 1.0f, 0.0f,
    //     -1.0f,  -1.0f, 0.0f,  0.0f, 0.0f, 1.0f, 0.0f, 0.0f,
    //     -1.0f, 1.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 1.0f

    //};

    float vertices[] = {
        // positions         // colors
         1.0f, 1.0f, 0.0f,  1.0f, 0.0f, 0.0f,  1.0f, 1.0f,
         1.0f, -1.0f, 0.0f,  0.0f, 1.0f, 0.0f,  1.0f, 0.0f,
         -1.0f,  -1.0f, 0.0f,  0.0f, 0.0f, 1.0f, 0.0f, 0.0f,
         -1.0f, 1.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 1.0f

    };
    unsigned int indices[] = {
        0, 1, 3,
        1, 2, 3
    };

    unsigned int VBO, VAO, EBO;
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glGenBuffers(1, &EBO);
    // bind the Vertex Array Object first, then bind and set vertex buffer(s), and then configure vertex attributes(s).
    glBindVertexArray(VAO);

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);
    // position attribute
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)0);
    glEnableVertexAttribArray(0);
    // color attribute
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)(3 * sizeof(float)));
    glEnableVertexAttribArray(1);

    glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, 8 * sizeof(float), (void*)(6 * sizeof(float)));
    glEnableVertexAttribArray(2);

    unsigned int texture;
    glGenTextures(1, &texture);
    glBindTexture(GL_TEXTURE_2D, texture);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    //glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
    
    CopyTexture(&h_texture, &d_texture, size * 3, true);
    rayTrace(d_scene, SCR_WIDTH, SCR_HEIGHT, d_texture);
    CopyTexture(&h_texture, &d_texture, size * 3, false);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, 1, 1, 0, GL_RGB, GL_UNSIGNED_BYTE, h_texture);

    // render loop
    // -----------
    while (!glfwWindowShouldClose(window))
    {
        // input
        // -----
        processInput(window);

        // render
        // ------
        glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        glBindTexture(GL_TEXTURE_2D, texture);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB8, SCR_WIDTH, SCR_HEIGHT, 0, GL_RGB, GL_UNSIGNED_BYTE, h_texture);

        ourShader.use();

        // render the triangle
        glBindVertexArray(VAO);
        //glDrawArrays(GL_TRIANGLES, 0, 3);
        glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);

        // glfw: swap buffers and poll IO events (keys pressed/released, mouse moved etc.)
        // -------------------------------------------------------------------------------
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    

    // optional: de-allocate all resources once they've outlived their purpose:
    // ------------------------------------------------------------------------
    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &VBO);

    // glfw: terminate, clearing all previously allocated GLFW resources.
    // ------------------------------------------------------------------
    glfwTerminate();

    //// Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    /*cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }*/

    return 0;
}

// process all input: query GLFW whether relevant keys are pressed/released this frame and react accordingly
// ---------------------------------------------------------------------------------------------------------
void processInput(GLFWwindow* window)
{
    if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);
}


// glfw: whenever the window size changed (by OS or user resize) this callback function executes
// ---------------------------------------------------------------------------------------------
void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
    // make sure the viewport matches the new window dimensions; note that width and 
    // height will be significantly larger than specified on retina displays.
    glViewport(0, 0, width, height);
}

hipError_t fillWithCuda(unsigned char* table, int SCR_WIDTH, int SCR_HEIGHT)
{
    char* d_table;
    hipError_t cudaStatus;
    int size = SCR_HEIGHT * SCR_WIDTH * 3;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_table, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(d_table, table, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    dim3 blocks3(SCR_HEIGHT, SCR_WIDTH, 1);
    fillKernel << <blocks3, 1 >> > (d_table, SCR_WIDTH);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    dim3 block32(10, 400, 1);
    fillKernel2 << <block32, 1 >> > (d_table, SCR_WIDTH);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    cudaStatus = hipMemcpy(table, d_table, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


Error:
    hipFree(d_table);
    return cudaStatus;
}
