#include "cudaFunctions.h"

#include <stdio.h>
#include <iostream>

hipError_t fillWithCuda(char* table, int SCR_WIDTH, int SCR_HEIGHT)
{
    char* d_table;
    hipError_t cudaStatus;
    int size = SCR_HEIGHT * SCR_WIDTH * 3;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_table, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(d_table, table, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    dim3 blocks3(SCR_HEIGHT, SCR_WIDTH, 1);
    fillKernel << <blocks3, 1 >> > (d_table, SCR_WIDTH);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(table, d_table, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


Error:
    hipFree(d_table);
    return cudaStatus;
}